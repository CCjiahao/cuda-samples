#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#include "hip/hip_runtime_api.h"

namespace cg = cooperative_groups;


__global__ void clock_block(clock_t *d_output, clock_t clock_count) {
    clock_t start_clock = clock();
    clock_t clock_offset = 0;
    while (clock_offset < clock_count) {
        clock_t end_clock = clock();
        clock_offset = end_clock - start_clock;
    }
    d_output[0] = clock_offset;
}

template <typename T>
__inline__ __device__ T warpSumReduce(T value) {
    value += __shfl_down_sync(0xFFFFFFFF, value , 16);
    value += __shfl_down_sync(0xFFFFFFFF, value , 8);
    value += __shfl_down_sync(0xFFFFFFFF, value , 4);
    value += __shfl_down_sync(0xFFFFFFFF, value , 2);
    value += __shfl_down_sync(0xFFFFFFFF, value , 1);
    return value;
}

__global__ void sum(clock_t *d_clocks, int N) {
    cg::thread_block cta = cg::this_thread_block();

    __shared__ int shared[32];

    int value = 0;
    for (int i = threadIdx.x; i < N; i += blockDim.x) {
        value += d_clocks[i];
    }
    value = warpSumReduce<int>(value);

    int laneId = threadIdx.x % warpSize;
    int warpId = threadIdx.x / warpSize;

    if (laneId == 0) shared[warpId] = value;
    cta.sync();
    value = (threadIdx.x < blockDim.x / warpSize) ? shared[laneId] : 0;     // assert blockDim.x % warpSize == 0
    if(warpId == 0) value = warpSumReduce<int>(value);

    if(threadIdx.x == 0) d_clocks[blockIdx.x] = value;
}

int main(int argc, char* argv[]) {
    // hyp
    int nkernels = 64;
    int nstreams = nkernels + 1;
    int nbytes = nkernels * sizeof(clock_t);
    float kernel_time = 10;

    // get device & info
    int devID = findCudaDevice(argc, (const char**)argv);
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
    if (deviceProp.concurrentKernels == 0) {
        printf("> GPU does not support concurrent kernel execution\n");
        printf("  CUDA kernel runs will be serialized\n");
    }
    printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // create stream
    hipStream_t* streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int i = 0; i < nstreams; i++) checkCudaErrors(hipStreamCreate(&streams[i]));

    // create event
    hipEvent_t start_event, stop_event;
    checkCudaErrors(hipEventCreate(&start_event));
    checkCudaErrors(hipEventCreate(&stop_event));
    hipEvent_t* events = (hipEvent_t*) malloc(nkernels * sizeof(hipEvent_t));
    for(int i = 0; i < nkernels; i++) checkCudaErrors(hipEventCreateWithFlags(&events[i], hipEventDisableTiming));

    // malloc d_output
    clock_t *d_output;
    checkCudaErrors(hipMalloc((void **)&d_output, nbytes));
    
    // time
    clock_t time_clocks = (clock_t)(kernel_time * deviceProp.clockRate);
    clock_t total_clocks = time_clocks * nkernels;
    
    // work
    hipEventRecord(start_event, 0);
    for (int i = 0; i < nkernels; ++i) {
        clock_block<<<1, 1, 0, streams[i]>>>(&d_output[i], time_clocks);
        checkCudaErrors(hipEventRecord(events[i], streams[i]));
        checkCudaErrors(hipStreamWaitEvent(streams[nstreams - 1], events[i], 0));
    }
    clock_t* output = (clock_t*)malloc(nbytes);
    checkCudaErrors(hipMemcpyAsync(output, d_output, nbytes, hipMemcpyDeviceToHost, streams[nstreams - 1]));
    sum<<<1, min((nkernels + 31) / 32 * 32, 1024), 0, streams[nstreams - 1]>>>(d_output, nkernels);
    clock_t output_sum;
    checkCudaErrors(hipMemcpyAsync(&output_sum, d_output, sizeof(clock_t), hipMemcpyDeviceToHost, streams[nstreams - 1]));
    checkCudaErrors(hipEventRecord(stop_event, 0));
    checkCudaErrors(hipEventSynchronize(stop_event));

    // time
    float elapsed_time;
    checkCudaErrors(hipEventElapsedTime(&elapsed_time, start_event, stop_event));

    printf("Expected time for serial execution of %d kernels = %.3fs\n", nkernels, nkernels * kernel_time / 1000.0f);
    printf("Expected time for concurrent execution of %d kernels = %.3fs\n", nkernels, kernel_time / 1000.0f);
    printf("Measured time for sample = %.3fs\n", elapsed_time / 1000.0f);

    // check
    assert(output_sum > total_clocks);
    clock_t output_sum2 = 0;
    for(int i = 0; i < nkernels; i++) output_sum2 += output[i];
    assert(output_sum == output_sum2);

    // free
    for (int i = 0; i < nkernels; i++) {
        hipStreamDestroy(streams[i]);
        hipEventDestroy(events[i]);
    }
    hipStreamDestroy(streams[nkernels]);
    free(streams);
    free(events);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipFree(d_output);

    return 0;
}